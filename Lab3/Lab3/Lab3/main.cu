#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <fstream>
#include <iostream>
#include <string>
#include <iomanip>
#include <vector>
#include ""

using namespace std; 



class Item {
public:
	string Title;
	int Quantity;
	float Price;

	Item() {}

	Item(string parts[]) {
		this->Title = parts[0];
		this->Quantity = std::atoi(parts[1].c_str());
		this->Price = stof(parts[2].c_str());
	}

	string ToString() {
		char buff[100];
		snprintf(buff, sizeof(buff), "|%-20s|%-8d|%-7.2f\n", Title.c_str(), Quantity, Price);
		std::string buffAsStdStr = buff;
		return buffAsStdStr;
	}

	float calculateValue() {
		vector<char> bytes(Title.begin(), Title.end());
		int stringValues = 0;
		for (char i : bytes) {
			stringValues += i;
		}
		int temp = stringValues ^ Quantity;
		float finalV = temp * Price;
		return finalV;
	}

	int outputSize() {
		return Title.size() + 1 + to_string(calculateValue()).size();
	}
	
	string ToStringWithValue() {
		char buff[100];
		snprintf(buff, sizeof(buff), "|%s %f|\n", Title.c_str(), calculateValue());
		std::string buffAsStdStr = buff;
		return buffAsStdStr;
	}
};

class Items {
public:
	Item ItemArray[30];

	int size() {
		return sizeof(ItemArray) / sizeof(ItemArray[0]);
	}

	int maxCharSize(){
		int max = ItemArray[0].outputSize();
		for (int i = 1; i < size(); i++) {
			int isize = ItemArray[i].outputSize();
			max = isize > max ? isize : max;
		}
		return max + 1; //FORMATAS: 'TITLE-value '
	}
};

Items* readItems(string file) {
	auto items = new Items();
	string s;
	ifstream ifs(file);
	std::string delimiter = ";";
	int CurrentLine = 0;
	string itemsParsed[3];
	if (ifs.is_open())
	{
		while (getline(ifs, s))
		{
			int current = 0;
			size_t pos = 0;
			string token;
			while ((pos = s.find(delimiter)) != std::string::npos) {
				token = s.substr(0, pos);
				itemsParsed[current++] = token;
				//std::cout << token << std::endl;
				s.erase(0, pos + delimiter.length());
			}
			items->ItemArray[CurrentLine++] = Item(itemsParsed);
		}
		ifs.close();
	}
	else cout << "Unable to open file";
	return items;
}

__global__ void run_on_gpu(Item* ItemArray, char* results, int* size, unsigned int* count);
__device__ float calculateValue(Item* item);

int main() {
	int gijuKiekis = 7;
	string fileName = "Data/IFF8-12_AkramasJ_L1_dat_1.txt";	
	//---RAM kintamieji
	Items *items = readItems(fileName);
	int sector_size = items->maxCharSize();
	int resultSize = sizeof(char) * sector_size * 30;
	auto *results = malloc(resultSize);
	int size = items->size();
	unsigned int count = 0;
	//---VRAM kintamieji
	Item* cuda_items;
	char *cuda_results;
	int* cuda_size;
	unsigned int *cuda_count;
	//---
	hipMalloc(&cuda_items, sizeof(Items));
	hipMalloc(&cuda_results, resultSize);
	hipMalloc(&cuda_size, sizeof(int));
	hipMalloc(&cuda_count, sizeof(unsigned int));
	//---
	hipMemcpy(cuda_items, items->ItemArray, sizeof(Items), hipMemcpyHostToDevice);
	hipMemcpy(cuda_results, results, resultSize, hipMemcpyHostToDevice);
	hipMemcpy(cuda_size, &size, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuda_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice);
	//---
	run_on_gpu << <1, gijuKiekis >> > (cuda_items, cuda_results, cuda_size, cuda_count); //Paleidzia gijas
	//---
	hipDeviceSynchronize(); //Palaukti visu giju
	//---
	hipMemcpy(&count, cuda_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
	//---

	cout << count << endl;
	//Print results;

	//---
	delete(items);
	free(results);
	hipFree(cuda_items);
	hipFree(cuda_results);
	hipFree(cuda_size);
	hipFree(cuda_count);
	//---
	cout << "Finished" << endl;
}

__global__ void run_on_gpu(Item* ItemArray, char *results, int *size, unsigned int *count) {
	int slice_size = *size / blockDim.x;
	//---
	int start_index = slice_size * threadIdx.x;
	int end_index = (threadIdx.x == blockDim.x - 1)? *size : slice_size * (threadIdx.x + 1);
	//---
	for (int i = start_index; i < end_index; i++) {
		float result = calculateValue(&ItemArray[i]);
		if (result > 0.5f) {
			atomicAdd(count, 1);
		}
	}
}

__device__ float calculateValue(Item* item) {
	//---
	/*
	string Title = item->Title;
	int Quantity = item->Quantity;
	float Price = item->Price;
	//---
	vector<char> bytes(Title.begin(), Title.end());
	int stringValues = 0;
	for (char i : bytes) {
		stringValues += i;
	}
	int temp = stringValues ^ Quantity;
	float finalV = temp * Price;
	//---

	return finalV;
	*/
	return 0.6f;
}