#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <fstream>
#include <iostream>
#include <string>
#include <iomanip>
#include <vector>
#include ""

using namespace std; 



class Item {
public:
	string Title;
	int Quantity;
	float Price;

	Item() {}

	Item(string parts[]) {
		this->Title = parts[0];
		this->Quantity = std::atoi(parts[1].c_str());
		this->Price = stof(parts[2].c_str());
	}

	string ToString() {
		char buff[100];
		snprintf(buff, sizeof(buff), "|%-20s|%-8d|%-7.2f\n", Title.c_str(), Quantity, Price);
		std::string buffAsStdStr = buff;
		return buffAsStdStr;
	}

	float calculateValue() {
		vector<char> bytes(Title.begin(), Title.end());
		int stringValues = 0;
		for (char i : bytes) {
			stringValues += i;
		}
		int temp = stringValues ^ Quantity;
		float finalV = temp * Price;
		return finalV;
	}

	int outputSize() {
		return Title.size() + 1 + to_string(calculateValue()).size();
	}
	
	string ToStringWithValue() {
		char buff[100];
		snprintf(buff, sizeof(buff), "|%s %f|\n", Title.c_str(), calculateValue());
		std::string buffAsStdStr = buff;
		return buffAsStdStr;
	}
};

class Items {
public:
	Item ItemArray[30];

	int size() {
		return sizeof(ItemArray) / sizeof(ItemArray[0]);
	}

	int maxCharSize(){
		int max = ItemArray[0].outputSize();
		for (int i = 1; i < size(); i++) {
			int isize = ItemArray[i].outputSize();
			max = isize > max ? isize : max;
		}
		return max + 1; //FORMATAS: 'TITLE-value '
	}

	void parseData(string* title, int *quantity, float *price) {
		for (int i = 0; i < size(); i++) {
			title[i] = ItemArray[i].Title;
			quantity[i] = ItemArray[i].Quantity;
			price[i] = ItemArray[i].Price;
		}
	}
};

Items* readItems(string file) {
	auto items = new Items();
	string s;
	ifstream ifs(file);
	std::string delimiter = ";";
	int CurrentLine = 0;
	string itemsParsed[3];
	if (ifs.is_open())
	{
		while (getline(ifs, s))
		{
			int current = 0;
			size_t pos = 0;
			string token;
			while ((pos = s.find(delimiter)) != std::string::npos) {
				token = s.substr(0, pos);
				itemsParsed[current++] = token;
				//std::cout << token << std::endl;
				s.erase(0, pos + delimiter.length());
			}
			items->ItemArray[CurrentLine++] = Item(itemsParsed);
		}
		ifs.close();
	}
	else cout << "Unable to open file";
	return items;
}

__global__ void run_on_gpu(int* quantity, float* price, char* results, int* size, unsigned int* count);
__device__ float calculateValue(int quantity, float price);

int main() {
	int gijuKiekis = 7;
	string fileName = "Data/IFF8-12_AkramasJ_L1_dat_1.txt";	
	//---RAM kintamieji
	Items *items = readItems(fileName);
	int sector_size = items->maxCharSize();
	int size = items->size();
	//---
	string title[30];
	int quantity[30];
	float price[30];
	items->parseData(title, quantity, price);
	int resultSize = sizeof(char) * sector_size * size;
	auto *results = malloc(resultSize);
	unsigned int count = 0;
	//---VRAM kintamieji
	int* cuda_quantity;
	float* cuda_price;
	char *cuda_results;
	int* cuda_size;
	unsigned int *cuda_count;
	//---
	hipMalloc(&cuda_quantity, sizeof(int)*size);
	hipMalloc(&cuda_price, sizeof(float)*size);
	hipMalloc(&cuda_results, resultSize);
	hipMalloc(&cuda_size, sizeof(int));
	hipMalloc(&cuda_count, sizeof(unsigned int));
	//---
	hipMemcpy(cuda_quantity, quantity, sizeof(int) * size, hipMemcpyHostToDevice);
	hipMemcpy(cuda_price, price, sizeof(float) * size, hipMemcpyHostToDevice);
	hipMemcpy(cuda_results, results, resultSize, hipMemcpyHostToDevice);
	hipMemcpy(cuda_size, &size, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuda_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice);
	//---
	run_on_gpu << <1, gijuKiekis >> > (cuda_quantity, cuda_price, cuda_results, cuda_size, cuda_count); //Paleidzia gijas
	//---
	hipDeviceSynchronize(); //Palaukti visu giju
	//---
	hipMemcpy(&count, cuda_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
	//---

	cout << count << endl;
	//Print results;

	//---
	delete(items);
	free(results);
	hipFree(cuda_quantity);
	hipFree(cuda_price);
	hipFree(cuda_results);
	hipFree(cuda_size);
	hipFree(cuda_count);
	//---
	cout << "Finished" << endl;
}

__global__ void run_on_gpu(int* quantity, float* price, char *results, int *size, unsigned int *count) {
	int slice_size = *size / blockDim.x;
	//---
	int start_index = slice_size * threadIdx.x;
	int end_index = (threadIdx.x == blockDim.x - 1)? *size : slice_size * (threadIdx.x + 1);
	//---
	for (int i = start_index; i < end_index; i++) {
		float result = calculateValue(quantity[i], price[i]);
		if (result > 0.5f) {
			atomicAdd(count, 1);
		}
	}
}

__device__ float calculateValue(int quantity, float price) {
	//---
	/*
	vector<char> bytes(Title.begin(), Title.end());
	int stringValues = 0;
	for (char i : bytes) {
		stringValues += i;
	}
	int temp = stringValues ^ Quantity;
	float finalV = temp * Price;
	//---

	return finalV;
	*/
	return 0.6f;
}