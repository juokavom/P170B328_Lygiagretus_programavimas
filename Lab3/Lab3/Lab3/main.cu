#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <fstream>
#include <iostream>
#include <string>
#include <iomanip>
#include <algorithm>
#include <vector>
#include ""
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/zip_iterator.h>

using namespace std;
using namespace thrust;



class Item {
public:
	string Title;
	int Quantity;
	float Price;

	Item() {}

	Item(string parts[]) {
		this->Title = parts[0];
		this->Quantity = std::atoi(parts[1].c_str());
		this->Price = stof(parts[2].c_str());
	}

	string ToString() {
		char buff[100];
		snprintf(buff, sizeof(buff), "|%-20s|%-8d|%-7.2f\n", Title.c_str(), Quantity, Price);
		std::string buffAsStdStr = buff;
		return buffAsStdStr;
	}

	float calculateValue() {
		vector<char> bytes(Title.begin(), Title.end());
		int stringValues = 0;
		for (char i : bytes) {
			stringValues += i;
		}
		int temp = stringValues ^ Quantity;
		float finalV = temp * Price;
		return finalV;
	}

	int outputSize() {
		return Title.size() + 1 + to_string(calculateValue()).size();
	}

	string ToStringWithValue() {
		char buff[100];
		snprintf(buff, sizeof(buff), "|%s %f|\n", Title.c_str(), calculateValue());
		std::string buffAsStdStr = buff;
		return buffAsStdStr;
	}
};

class Items {
public:
	Item ItemArray[30];

	int size() {
		return sizeof(ItemArray) / sizeof(ItemArray[0]);
	}

	int maxCharSize() {
		int max = ItemArray[0].outputSize();
		for (int i = 1; i < size(); i++) {
			int isize = ItemArray[i].outputSize();
			max = isize > max ? isize : max;
		}
		return max + 1; //FORMATAS: 'TITLE-value '
	}

	void parseData(char* title, int* titleIndex, int* quantity, float* price, int* chunkSize) {
		for (int i = 0; i < size(); i++) {
			char* curTitle = &ItemArray[i].Title[0];
			titleIndex[i] = strlen(curTitle);
			for (int u = 0; u < titleIndex[i]; u++) {
				int globalIndex = i * *chunkSize + u;
				title[globalIndex] = curTitle[u];
			}
			for (int u = titleIndex[i]; u < *chunkSize; u++) {
				int globalIndex = i * *chunkSize + u;
				title[globalIndex] = ' ';
			}
			quantity[i] = ItemArray[i].Quantity;
			price[i] = ItemArray[i].Price;
		}
	}
};

Items* readItems(string file) {
	auto items = new Items();
	string s;
	ifstream ifs(file);
	std::string delimiter = ";";
	int CurrentLine = 0;
	string itemsParsed[3];
	if (ifs.is_open())
	{
		while (getline(ifs, s))
		{
			int current = 0;
			size_t pos = 0;
			string token;
			while ((pos = s.find(delimiter)) != std::string::npos) {
				token = s.substr(0, pos);
				itemsParsed[current++] = token;
				//std::cout << token << std::endl;
				s.erase(0, pos + delimiter.length());
			}
			items->ItemArray[CurrentLine++] = Item(itemsParsed);
		}
		ifs.close();
	}
	else cout << "Unable to open file";
	return items;
}

__global__ void run_on_gpu(char* title, int* titleLength, int* quantity, float* price, char* results, int* size, unsigned int* count, int* chunk);
__device__ float calculateValue(char* title, int titleLength, int quantity, float price);
__device__ char* getTitle(char* arr, int begin, int len);

int main() {
	int gijuKiekis = 7;
	string fileName = "Data/IFF8-12_AkramasJ_L1_dat_1.txt";
	//---RAM kintamieji
	Items* items = readItems(fileName);
	int arrayChunkSize = items->maxCharSize();
	int size = items->size();
	//---
	char* title = (char*)malloc(sizeof(char) * arrayChunkSize * size);
	int* titleLength = (int*)malloc(sizeof(int) * size);
	int* quantity = (int*)malloc(sizeof(int) * size);
	float* price = (float*)malloc(sizeof(float) * size);
	//---
	items->parseData(title, titleLength, quantity, price, &arrayChunkSize);
	int resultSize = sizeof(char) * arrayChunkSize * size;
	auto* results = malloc(resultSize);
	unsigned int count = 0;
	//---VRAM kintamieji
	char* cuda_title;
	int* cuda_title_length;
	int* cuda_quantity;
	float* cuda_price;
	char* cuda_results;
	int* cuda_size;
	unsigned int* cuda_count;
	int* cuda_chunk_size;
	//---
	hipMalloc(&cuda_title, sizeof(char) * arrayChunkSize * size);
	hipMalloc(&cuda_title_length, sizeof(int) * size);
	hipMalloc(&cuda_quantity, sizeof(int) * size);
	hipMalloc(&cuda_price, sizeof(float) * size);
	hipMalloc(&cuda_results, resultSize);
	hipMalloc(&cuda_size, sizeof(int));
	hipMalloc(&cuda_count, sizeof(unsigned int));
	hipMalloc(&cuda_chunk_size, sizeof(int));
	//---
	hipMemcpy(cuda_title, title, sizeof(char) * arrayChunkSize * size, hipMemcpyHostToDevice);
	hipMemcpy(cuda_title_length, titleLength, sizeof(int) * size, hipMemcpyHostToDevice);
	hipMemcpy(cuda_quantity, quantity, sizeof(int) * size, hipMemcpyHostToDevice);
	hipMemcpy(cuda_price, price, sizeof(float) * size, hipMemcpyHostToDevice);
	hipMemcpy(cuda_results, results, resultSize, hipMemcpyHostToDevice);
	hipMemcpy(cuda_size, &size, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuda_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(cuda_chunk_size, &arrayChunkSize, sizeof(int), hipMemcpyHostToDevice);
	//---
	run_on_gpu << <1, gijuKiekis >> > (cuda_title, cuda_title_length, cuda_quantity, cuda_price, cuda_results, cuda_size, cuda_count, cuda_chunk_size); //Paleidzia gijas
	//---
	hipDeviceSynchronize(); //Palaukti visu giju
	//---
	hipMemcpy(&count, cuda_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
	//---

	cout << count << endl;
	//Print results;

	//---
	delete(items);
	free(title);
	free(titleLength);
	free(quantity);
	free(price);
	free(results);
	hipFree(cuda_title);
	hipFree(cuda_title_length);
	hipFree(cuda_quantity);
	hipFree(cuda_price);
	hipFree(cuda_results);
	hipFree(cuda_size);
	hipFree(cuda_count);
	hipFree(cuda_chunk_size);
	//---
	cout << "Finished" << endl;
}

__global__ void run_on_gpu(char* title, int* titleLength, int* quantity, float* price, char* results, int* size, unsigned int* count, int* chunk) {

	int slice_size = *size / blockDim.x;
	//---
	int start_index = slice_size * threadIdx.x;
	int end_index = (threadIdx.x == blockDim.x - 1) ? *size : slice_size * (threadIdx.x + 1);
	//---
	for (int i = start_index; i < end_index; i++) {
		int stringIndex = *chunk * i;
		int stringLength = titleLength[i];
		char* curr_title = getTitle(title, stringIndex, stringLength);
		float result = calculateValue(curr_title, titleLength[i], quantity[i], price[i]);
		result = result - (int)result;
		if (result > 0.5f) {
			atomicAdd(count, 1);
		}
	}
}
__device__ char* getTitle(char* arr, int begin, int len) {
	char* res = new char[len];
	for (int i = 0; i < len; i++) {
		res[i] = *(arr + begin + i);
	}
	return res;
}
__device__ float calculateValue(char* title, int titleLength, int quantity, float price) {
	int stringValues = 0;
	for (int i = 0; i < titleLength; i++) {
		stringValues += title[i];
	}
	int temp = stringValues ^ quantity;
	float finalV = temp * price;
	return finalV;
}