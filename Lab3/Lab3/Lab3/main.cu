#include "hip/hip_runtime.h"

#include <cstdio>
#include <fstream>
#include <iostream>
#include <string>
#include <iomanip>
#include <vector>


using namespace std; 


__global__ void run_on_gpu();
__device__ void execute(const char* name);

class Item {
public:
	string Title;
	int Quantity;
	float Price;

	Item() {}

	Item(string parts[]) {
		this->Title = parts[0];
		this->Quantity = std::atoi(parts[1].c_str());
		this->Price = stof(parts[2].c_str());
	}

	string ToString() {
		char buff[100];
		snprintf(buff, sizeof(buff), "|%-20s|%-8d|%-7.2f\n", Title.c_str(), Quantity, Price);
		std::string buffAsStdStr = buff;
		return buffAsStdStr;
	}
};

class Items {
public:
	Item ItemArray[30];

	int size() {
		return sizeof(ItemArray) / sizeof(ItemArray[0]);
	}
};

Items* readItems(string file) {
	auto items = new Items();
	string s;
	ifstream ifs(file);
	std::string delimiter = ";";
	int CurrentLine = 0;
	string itemsParsed[3];
	if (ifs.is_open())
	{
		while (getline(ifs, s))
		{
			int current = 0;
			size_t pos = 0;
			string token;
			while ((pos = s.find(delimiter)) != std::string::npos) {
				token = s.substr(0, pos);
				itemsParsed[current++] = token;
				//std::cout << token << std::endl;
				s.erase(0, pos + delimiter.length());
			}
			items->ItemArray[CurrentLine++] = Item(itemsParsed);
		}
		ifs.close();
	}
	else cout << "Unable to open file";
	return items;
}

int main() {
	int gijuKiekis = 7;
	string fileName = "Data/IFF8-12_AkramasJ_L1_dat_1.txt";	
	auto items = readItems(fileName);
	cout << sizeof(Items) << endl;
	/*
	run_on_gpu << <1, gijuKiekis >> > (); //Paleidzia gijas
	cudaDeviceSynchronize(); //Palaukti visu giju
	*/
	//Isvesti rezultatus


	delete(items);
	cout << "Finished" << endl;
}

__global__ void run_on_gpu() {
	const char* name;
	
	execute(name);
}

__device__ void execute(const char* name) {
	printf("%s: first\n", name);
	printf("%s: second\n", name);
	printf("%s: third\n", name);
}
