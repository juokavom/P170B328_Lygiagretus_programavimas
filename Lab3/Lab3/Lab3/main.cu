#include "hip/hip_runtime.h"

#include <cstdio>
#include <fstream>
#include <iostream>
#include <string>
#include <iomanip>
#include <vector>


using namespace std; 


//__global__ void run_on_gpu(Items* data, Items* results, int* size);
//__device__ void execute();

class Item {
public:
	string Title;
	int Quantity;
	float Price;

	Item() {}

	Item(string parts[]) {
		this->Title = parts[0];
		this->Quantity = std::atoi(parts[1].c_str());
		this->Price = stof(parts[2].c_str());
	}

	string ToString() {
		char buff[100];
		snprintf(buff, sizeof(buff), "|%-20s|%-8d|%-7.2f\n", Title.c_str(), Quantity, Price);
		std::string buffAsStdStr = buff;
		return buffAsStdStr;
	}

	float calculateValue() {
		vector<char> bytes(Title.begin(), Title.end());
		int stringValues = 0;
		for (char i : bytes) {
			stringValues += i;
		}
		int temp = stringValues ^ Quantity;
		float finalV = temp * Price;
		return finalV;
	}

	int outputSize() {
		return Title.size() + 1 + to_string(calculateValue()).size();
	}
	
	string ToStringWithValue() {
		char buff[100];
		snprintf(buff, sizeof(buff), "|%s %f|\n", Title.c_str(), calculateValue());
		std::string buffAsStdStr = buff;
		return buffAsStdStr;
	}
};

class Items {
public:
	Item ItemArray[30];

	int size() {
		return sizeof(ItemArray) / sizeof(ItemArray[0]);
	}

	int maxCharSize(){
		int max = ItemArray[0].outputSize();
		for (int i = 1; i < size(); i++) {
			int isize = ItemArray[i].outputSize();
			max = isize > max ? isize : max;
		}
		return max + 1; //FORMATAS: 'TITLE-value '
	}
};

Items* readItems(string file) {
	auto items = new Items();
	string s;
	ifstream ifs(file);
	std::string delimiter = ";";
	int CurrentLine = 0;
	string itemsParsed[3];
	if (ifs.is_open())
	{
		while (getline(ifs, s))
		{
			int current = 0;
			size_t pos = 0;
			string token;
			while ((pos = s.find(delimiter)) != std::string::npos) {
				token = s.substr(0, pos);
				itemsParsed[current++] = token;
				//std::cout << token << std::endl;
				s.erase(0, pos + delimiter.length());
			}
			items->ItemArray[CurrentLine++] = Item(itemsParsed);
		}
		ifs.close();
	}
	else cout << "Unable to open file";
	return items;
}

int main() {
	int gijuKiekis = 7;
	string fileName = "Data/IFF8-12_AkramasJ_L1_dat_1.txt";	
	//---RAM kintamieji
	Items *items = readItems(fileName);
	cout << items->maxCharSize();
	//string *results = new Items();
	int size = items->size();
	int count = 0;
	//---VRAM kintamieji
	Items *cuda_items, *cuda_results;
	int *cuda_size, *cuda_count;
	//---
	/*
	cudaMalloc(&cuda_items, sizeof(Items));
	cudaMalloc(&cuda_results, sizeof(Items));
	cudaMalloc(&cuda_size, sizeof(int));
	cudaMalloc(&cuda_count, sizeof(int));
	//---
	cudaMemcpy(cuda_items, items, sizeof(Items), cudaMemcpyHostToDevice);
	cudaMemcpy(cuda_results, results, sizeof(Items), cudaMemcpyHostToDevice);
	cudaMemcpy(cuda_size, &size, sizeof(int), cudaMemcpyHostToDevice);
	cudaMemcpy(cuda_count, &count, sizeof(int), cudaMemcpyHostToDevice);
	//---
	run_on_gpu << <1, gijuKiekis >> > (cuda_items, cuda_results, cuda_size); //Paleidzia gijas
	//---
	cudaDeviceSynchronize(); //Palaukti visu giju
	//---
	cudaMemcpy(results, cuda_results, sizeof(Items), cudaMemcpyDeviceToHost);
	//---

	//Print results;

	//---
	delete(items);
	delete(results);
	cudaFree(cuda_items);
	cudaFree(cuda_results);
	cudaFree(cuda_size);
	*/
	//---
	cout << "Finished" << endl;
}

/*
__global__ void run_on_gpu(Items *data, Items *results, int *size) {
	int slice_size = *size / blockDim.x;
	//---
	int start_index = slice_size * threadIdx.x;
	int end_index = (threadIdx.x == blockDim.x - 1)? *size : slice_size * (threadIdx.x + 1);
	//---

	execute();
}

__device__ void execute() {
	printf("%s: first\n");
	printf("%s: second\n");
	printf("%s: third\n");
}
*/