#include "hip/hip_runtime.h"

#include <cstdio>
//#include "nlohmann.hpp"
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <string>


using namespace std;
//using json = nlohmann::json;

__global__ void run_on_gpu();
__device__ void execute(const char* name);

class Item {
public:
    string Title;
    int Quantity;
    float Price;

    Item() {}

    Item(string title, int quantity, float price) {
        this->Title = title;
        this->Quantity = quantity;
        this->Price = price;
    }

    string ToString() {
        char buff[100];
        snprintf(buff, sizeof(buff), "|%-20s|%-8d|%-7.2f", Title.c_str(), Quantity, Price);
        std::string buffAsStdStr = buff;
        return buffAsStdStr;
    }
};

class Items {
public:
    Item ItemArray[30];

    int size() {
        return sizeof(ItemArray) / sizeof(ItemArray[0]);
    }
};

int main() {
    string line;
    ifstream myfile("Data/IFF8-12_AkramasJ_L1_dat_1.json");
    if (myfile.is_open())
    {
        while (getline(myfile, line))
        {
            cout << line << '\n';
        }
        myfile.close();
    }
    else cout << "Unable to open file";
    /*
    run_on_gpu << <1, 2 >> > ();
    cudaDeviceSynchronize();
    */
    cout << "Finished" << endl;
}

__global__ void run_on_gpu() {
    const char* name;
    if (threadIdx.x == 0) {
        name = "Thread 1";
    }
    else {
        name = "Thread 2";
    }
    execute(name);
}

__device__ void execute(const char* name) {
    printf("%s: first\n", name);
    printf("%s: second\n", name);
    printf("%s: third\n", name);
}
