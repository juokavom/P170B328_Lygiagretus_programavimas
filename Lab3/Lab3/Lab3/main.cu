#include "hip/hip_runtime.h"
//Jokubas Akramas IFF-8/12
//3 laboratorinis darbas
//Lygiagretus progrmavimas
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <fstream>
#include <iostream>
#include <string>

using namespace std;

void writeToFile(char* fileName, char* results, int n);
__global__ void runOnGPU(char* title, int* titleLength, int* quantity, float* price, char* results, int* size, unsigned int* count, int* chunk);
__device__ float calculateValue(char* title, int titleLength, int quantity, float price);
__device__ char* getTitle(char* arr, int begin, int len);
__device__ void writeItem(char* results, unsigned int* count, int* chunk, char* title, int titleLength, float* result, int* res);

class Item {
public:
	string Title;
	int Quantity;
	float Price;

	Item() {}

	Item(string parts[]) {
		this->Title = parts[0];
		this->Quantity = std::atoi(parts[1].c_str());
		this->Price = stof(parts[2].c_str());
	}

	int outputSize() {
		return Title.size() + 3;
	}
};

class Items {
public:
	Item ItemArray[30];

	int size() {
		return sizeof(ItemArray) / sizeof(ItemArray[0]);
	}

	int maxCharSize() {
		int max = ItemArray[0].outputSize();
		for (int i = 1; i < size(); i++) {
			int isize = ItemArray[i].outputSize();
			max = isize > max ? isize : max;
		}
		return max + 1; //FORMATAS: 'TITLE-XX '
	}

	void parseData(char* title, int* titleIndex, int* quantity, float* price, int* chunkSize) {
		for (int i = 0; i < size(); i++) {
			char* curTitle = &ItemArray[i].Title[0];
			titleIndex[i] = strlen(curTitle);
			for (int u = 0; u < titleIndex[i]; u++) {
				int globalIndex = i * *chunkSize + u;
				title[globalIndex] = curTitle[u];
			}
			for (int u = titleIndex[i]; u < *chunkSize; u++) {
				int globalIndex = i * *chunkSize + u;
				title[globalIndex] = ' ';
			}
			quantity[i] = ItemArray[i].Quantity;
			price[i] = ItemArray[i].Price;
		}
	}
};

Items* readItems(string file) {
	auto items = new Items();
	string s;
	ifstream ifs(file);
	std::string delimiter = ";";
	int CurrentLine = 0;
	string itemsParsed[3];
	if (ifs.is_open())
	{
		while (getline(ifs, s))
		{
			int current = 0;
			size_t pos = 0;
			string token;
			while ((pos = s.find(delimiter)) != std::string::npos) {
				token = s.substr(0, pos);
				itemsParsed[current++] = token;
				s.erase(0, pos + delimiter.length());
			}
			items->ItemArray[CurrentLine++] = Item(itemsParsed);
		}
		ifs.close();
	}
	else cout << "Negalima atidaryti failo";
	return items;
}


int main() {
	int gijuKiekis = 7;
	string fileName = "Data/IFF8-12_AkramasJ_L3_dat_1.txt";
	//---RAM kintamieji
	Items* items = readItems(fileName);
	int arrayChunkSize = items->maxCharSize();
	int size = items->size();
	unsigned int count = 0;
	//---
	char* title = (char*)malloc(sizeof(char) * arrayChunkSize * size);
	int* titleLength = (int*)malloc(sizeof(int) * size);
	int* quantity = (int*)malloc(sizeof(int) * size);
	float* price = (float*)malloc(sizeof(float) * size);
	//---
	items->parseData(title, titleLength, quantity, price, &arrayChunkSize);
	//---VRAM kintamieji
	char* cuda_title, * cuda_results;
	int* cuda_title_length, * cuda_quantity, * cuda_size, * cuda_chunk_size;
	float* cuda_price;
	unsigned int* cuda_count;
	//---
	hipMalloc(&cuda_title, sizeof(char) * arrayChunkSize * size);
	hipMalloc(&cuda_title_length, sizeof(int) * size);
	hipMalloc(&cuda_quantity, sizeof(int) * size);
	hipMalloc(&cuda_price, sizeof(float) * size);
	hipMalloc(&cuda_results, sizeof(char) * arrayChunkSize * size);
	hipMalloc(&cuda_size, sizeof(int));
	hipMalloc(&cuda_count, sizeof(unsigned int));
	hipMalloc(&cuda_chunk_size, sizeof(int));
	//---
	hipMemcpy(cuda_title, title, sizeof(char) * arrayChunkSize * size, hipMemcpyHostToDevice);
	hipMemcpy(cuda_title_length, titleLength, sizeof(int) * size, hipMemcpyHostToDevice);
	hipMemcpy(cuda_quantity, quantity, sizeof(int) * size, hipMemcpyHostToDevice);
	hipMemcpy(cuda_price, price, sizeof(float) * size, hipMemcpyHostToDevice);
	hipMemcpy(cuda_size, &size, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuda_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(cuda_chunk_size, &arrayChunkSize, sizeof(int), hipMemcpyHostToDevice);
	//---
	runOnGPU << <1, gijuKiekis >> > (cuda_title, cuda_title_length, cuda_quantity, cuda_price, cuda_results, cuda_size, cuda_count, cuda_chunk_size); //Paleidzia gijas
	//---
	hipDeviceSynchronize(); //Palaukti visu giju
	//---
	hipMemcpy(&count, cuda_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
	char* results = (char*)malloc(sizeof(char) * arrayChunkSize * count);
	hipMemcpy(results, cuda_results, sizeof(char) * arrayChunkSize * count, hipMemcpyDeviceToHost);
	//---
	writeToFile("Data/IFF8-12_AkramasJ_L3_rez.txt", results, sizeof(char) * arrayChunkSize * count);
	//---
	delete(items);
	//---
	free(title);
	free(titleLength);
	free(quantity);
	free(price);
	free(results);
	//---
	hipFree(cuda_title);
	hipFree(cuda_title_length);
	hipFree(cuda_quantity);
	hipFree(cuda_price);
	hipFree(cuda_results);
	hipFree(cuda_size);
	hipFree(cuda_count);
	hipFree(cuda_chunk_size);
}

void writeToFile(char* fileName, char* results, int n) {
	ofstream myfile(fileName);
	if (myfile.is_open()) {
		for (int i = 0; i < n; i++) {
			myfile << results[i];
			cout << results[i];
		}
		myfile.close();
	}
	else cout << "Unable to open file";
}

__global__ void runOnGPU(char* title, int* titleLength, int* quantity, float* price, char* results, int* size, unsigned int* count, int* chunk) {
	int slice_size = *size / blockDim.x;
	//---
	int start_index = slice_size * threadIdx.x;
	int end_index = (threadIdx.x == blockDim.x - 1) ? *size : slice_size * (threadIdx.x + 1);
	//---
	for (int i = start_index; i < end_index; i++) {
		int stringIndex = *chunk * i;
		int stringLength = titleLength[i];
		char* curr_title = getTitle(title, stringIndex, stringLength);
		float result = calculateValue(curr_title, titleLength[i], quantity[i], price[i]);
		float result2 = result - (int)result;
		if (result2 > 0.5f) {
			int res = result2 * 100;
			unsigned int current_count = atomicAdd(count, 1);
			writeItem(results, &current_count, chunk, curr_title, titleLength[i], &result, &res);
		}
	}
}
__device__ void writeItem(char* results, unsigned int* count, int* chunk, char* title, int titleLength, float* result, int* res) {
	int current_index = (int)*count * *chunk;
	int end_index = current_index + *chunk;
	//---
	for (int i = 0; i < titleLength; i++) {
		char titleValue = (title[i] >= 97 && title[i] <= 122) ? title[i] - 32 : title[i];
		titleValue = (title[i] == ' ') ? '_' : titleValue;
		results[current_index] = titleValue;
		current_index++;
	}
	//---
	results[current_index] = '-';
	current_index++;
	//---
	int nr = *res / 10;
	results[current_index] = ('0' + nr);
	current_index++;
	//---
	nr = *res - nr * 10;
	results[current_index] = ('0' + nr);
	current_index++;
	//---
	for (int i = current_index; i < end_index; i++) {
		results[i] = ' ';
	}
}
__device__ char* getTitle(char* arr, int begin, int len) {
	char* res = new char[len];
	for (int i = 0; i < len; i++) {
		res[i] = *(arr + begin + i);
	}
	return res;
}
__device__ float calculateValue(char* title, int titleLength, int quantity, float price) {
	int stringValues = 0;
	for (int i = 0; i < titleLength; i++) {
		stringValues += title[i];
	}
	int temp = stringValues ^ quantity;
	float finalV = temp * price;
	return finalV;
}